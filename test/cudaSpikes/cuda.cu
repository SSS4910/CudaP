#include "hip/hip_runtime.h"

#include "cuda.cuh"
#include "book.h"


// Declare the Cuda kernels and any Cuda functions
__global__ void analyze_id(Buffer *buffer, int *statResults)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {

    }
    
}

__global__ void analyze_age(Buffer *buffer, int *statResults)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {

    }

}

__global__ void analyze_height(Buffer *buffer, int *statResults)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {

    }
    
}

int analyze_data(Buffer *input_buffer)
{
    int threads = N;
    int blocks = threads/2; // to be more conservative w/blocks: (N + (threads - 1)) / threads
    
    if(blocks > MAX_BLOCKS)
    {
        printf("Data set is too large\nSet cannot exceed %d elements\n", MAX_BLOCKS);
        exit(1);
    }

    cuda_setup(COMPUTE_ABILITY);

    // CPU variables
    Buffer *cudaBuffer;
    int idStats[N];
    int ageStats[N];
    int heightStats[N];

    // GPU variables
    Buffer *dev_buffer;
    int *dev_idStats;
    int *dev_ageStats;
    int *dev_heightStats;

    // events to track performance time
    float elapsedTime;
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // creates zero-copy memory for buffer (both CPU and GPU point to same memory). A pointer will be given to the GPU later...
    HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer, sizeof(input_buffer), hipHostMallocWriteCombined | hipHostMallocMapped));

    // fill the zero-copy memorywith data from input_buffer
    cudaBuffer = (Buffer *)memcpy(&cudaBuffer, &input_buffer, sizeof(input_buffer));

    // lets main know the buffer is free
    input_buffer->available = TRUE;

    //Declare streams
    hipStream_t stream0;
    hipStream_t stream1;
    hipStream_t stream2;

    //create steams
    HANDLE_ERROR(hipStreamCreate(&stream0));
    HANDLE_ERROR(hipStreamCreate(&stream1));
    HANDLE_ERROR(hipStreamCreate(&stream2));

    // allocating GPU memory (GPU only memory)
    HANDLE_ERROR(hipMalloc( (void **) &dev_idStats, N * sizeof(int) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_ageStats, N * sizeof(int) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_heightStats, N * sizeof(int) ));

    // page-locking output buffers (pin host memory for streams)
    HANDLE_ERROR(hipHostAlloc( (void **) &idStats, N * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc( (void **) &ageStats, N * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc( (void **) &heightStats, N * sizeof(int), hipHostMallocDefault));

    /* FILL BUFFER WITH DATA */

    // gives a pointer to the GPU to reference the zero-copy memory
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_buffer, cudaBuffer, 0));

    // calls to Cuda kernels, note streams have been added
    analyze_id<<< blocks, threads, 0, stream0 >>>(dev_buffer, dev_idStats);
    analyze_age<<< blocks, threads, 0, stream1 >>>(dev_buffer, dev_ageStats);
    analyze_height<<< blocks, threads, 0, stream2 >>>(dev_buffer, dev_heightStats);

    // Get the results from the GPU
    HANDLE_ERROR(hipMemcpyAsync(idStats, dev_idStats, N * sizeof(int), hipMemcpyDeviceToHost, stream0));
    HANDLE_ERROR(hipMemcpyAsync(ageStats, dev_ageStats, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
    HANDLE_ERROR(hipMemcpyAsync(heightStats, dev_heightStats, N * sizeof(int), hipMemcpyDeviceToHost, stream2));

    // make sure everyone is done
    HANDLE_ERROR(hipStreamSynchronize(stream0));
    HANDLE_ERROR(hipStreamSynchronize(stream1));
    HANDLE_ERROR(hipStreamSynchronize(stream2));

    HANDLE_ERROR(hipDeviceSynchronize());

    //stop timing events
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    // cuda cleanup
    hipFree(dev_buffer);
    hipFree(dev_idStats);
    hipFree(dev_ageStats);
    hipFree(dev_heightStats);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    // total results
    int idTotal = 0;
    int ageTotal = 0;
    int heightTotal = 0;

    int x = 0; // loop iterator

    // id totals
    for(x = 0; x < N; x++)
    {
        idTotal += idStats[x];
    }

    // age totals
    for(x = 0; x < N; x++)
    {
        ageTotal += ageStats[x];
    }

    // height totals
    for(x = 0; x < N; x++)
    {
        heightTotal += heightStats[x];
    }

    // print final totals
    printf("\n");
    printf("ID total: %d\n", idTotal);
    printf("Age total: %d\n", ageTotal);
    printf("Height total: %d\n", heightTotal);
    printf("\n");
    printf("blocks used: %d\n", blocks);
    printf("threads used: %d\n", threads);
    printf("threads per block: %d\n", (threads/blocks));
    printf("\n");
    printf("Elapsed time: %3.1f ms\n", elapsedTime);
    printf("\n");
    printf("End of cuda struct test\n");


    return 0;
}

int cuda_setup(int computeCapability)
{
    int dev;
    //int numOfDevs;

    hipDeviceProp_t prop;

    // assign number of cuda devices to 3rd element in specs array
    //HANDLE_ERROR(hipGetDeviceCount(&numOfDevs));

    // gets current device
    HANDLE_ERROR(hipGetDevice(&dev));

    // create a "pseudo" device w/ desired values and let cuda api pick device that matches
    // memset is just zero-ing out the specified memory
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = computeCapability;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));

    // sets the device w/ the desired paramaters as the device to use
    HANDLE_ERROR(hipSetDevice(dev));

    // allow GPU to zero-copy memory (map)
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

    // get more info about the device
    //HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));

    return 0;
}