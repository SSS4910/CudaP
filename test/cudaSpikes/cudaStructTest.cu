#include "hip/hip_runtime.h"
/*
    cudaStructTest
    testing/optimizing how to access/manipulate/return
    structures in cuda
*/

#include <stdio.h>
#include <unistd.h>

#include "book.h"

#define N 10
#define BLOCKS 1
#define THREADS 10

typedef struct{
    int id;
    int age;
    int height;
} Person;


__global__ void analyze_id(Person *people, int *statResults)
{

}

__global__ void analyze_age(Person *people, int *statResults)
{
    
}

__global__ void analyze_height(Person *people, int *statResults)
{
    
}


int main(void)
{
    // CPU variables
    Person *people;
    int *idStats[N];
    int *ageStats[N];
    int *heightStats[N];

    // GPU variables
    Person *dev_people;
    int *dev_idStats;
    int *dev_ageStats;
    int *dev_heightStats;

    // creates zero-copy memory for buffer (both CPU and GPU point to same memory). A pointer will be given to the GPU later...
    HANDLE_ERROR( hipHostAlloc( (void **) &people, N * sizeof(Person), hipHostMallocWriteCombined | hipHostMallocMapped));

    // allocating GPU memory (GPU only memory)
    HANDLE_ERROR(hipMalloc( (void **) &dev_idStats, N * sizeof(int) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_ageStats, N * sizeof(int) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_heightStats, N * sizeof(int) ));

    // fill people with some...
    Person person1  = {1, 25, 6};
    Person person2  = {2, 25, 5};
    Person person3  = {3, 25, 6};
    Person person4  = {4, 26, 5};
    Person person5  = {5, 26, 6};
    Person person6  = {6, 26, 5};
    Person person7  = {7, 27, 6};
    Person person8  = {8, 27, 5};
    Person person9  = {9, 27, 6};
    Person person10 = {10, 28, 5};

    people[0] = person1;
    people[1] = person2;
    people[2] = person3;
    people[3] = person4;
    people[4] = person5;
    people[5] = person6;
    people[6] = person7;
    people[7] = person8;
    people[8] = person9;
    people[9] = person10;

    // check data in people
    int x;
    for(x = 0; x < N; x++)
    {
        printf("id = %d\n", people[x].id);
        printf("age = %d\n", people[x].age);
        printf("height = %d\n", people[x].height);
        printf("\n");
    }

    // gives a pointer to the GPU to reference the zero-copy memory
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_people, people, 0));

    // calls to cuda kernels
    analyze_id<<<BLOCKS, THREADS>>>(dev_people, dev_idStats);
    analyze_age<<<BLOCKS, THREADS>>>(dev_people, dev_ageStats);
    analyze_height<<<BLOCKS, THREADS>>>(dev_people, dev_heightStats);

    // Get the results from the GPU
    HANDLE_ERROR(hipMemcpy(idStats, &dev_idStats, N * sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(ageStats, &dev_ageStats, N * sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(heightStats, &dev_heightStats, N * sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipDeviceSynchronize());

    return 0;


}

