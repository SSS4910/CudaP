#include "hip/hip_runtime.h"
/*
    cudaStructTest
    testing/optimizing how to access/manipulate/return
    structures in cuda
*/

#include <stdio.h>
#include <unistd.h>

#define N 10

typedef struct{
    int id;
    int age;
    int height;
} Person;

Person * populate_array(Person *people);

__global__ void analyze_id()
{

}

__global__ void analyze_age()
{
    
}

__global__ void analyze_height()
{
    
}


int main(void)
{
    Person *people = malloc(N * sizeof(Person));
    people = populate_array(people);

    return 0;


}

Person * populate_array(Person *people)
{
    Person person1;
    person1.id = 1;
    person1.age = 25;
    person1.height = 6;

    Person person2;
    person2.id = 2;
    person2.age = 21;
    person2.height = 5;

    Person person3;
    person3.id = 3;
    person3.age = 26;
    person3.height = 6;

    Person person4;
    person4.id = 4;
    person4.age = 23;
    person4.height = 5;

    Person person5;
    person5.id = 5;
    person5.age = 25;
    person5.height = 7;

    Person person6;
    person6.id = 6;
    person6.age = 27;
    person6.height = 5;

    Person person7;
    person7.id = 7;
    person7.age = 29;
    person7.height = 6;

    Person person8;
    person8.id = 8;
    person8.age = 24;
    person8.height = 6;

    Person person9;
    person9.id = 9;
    person9.age = 27;
    person9.height = 6;

    Person person10;
    person10.id = 10;
    person10.age = 30;
    person10.height = 2;

    people[0] = person1;
    people[1] = person2;
    people[2] = person3;
    people[3] = person4;
    people[4] = person5;
    people[5] = person6;
    people[6] = person7;
    people[7] = person8;
    people[8] = person9;
    people[9] = person10;

    return people;

}