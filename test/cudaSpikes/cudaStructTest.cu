#include "hip/hip_runtime.h"
/*
    cudaStructTest
    testing/optimizing how to access/manipulate/return
    structures in cuda.
*/

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

#include "book.h"

#define N 30
#define TRUE 1
#define FALSE 0
#define MAX_BLOCKS 65000
/*#define BLOCKS 2
#define THREADS 5*/

int cuda_setup(int computeCapability);

typedef struct{
    int id;
    int age;
    int height;
} Person;


// Declare the Cuda kernels and any Cuda functions
__global__ void analyze_id(Person *people, int *statResults)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {
        Person person = people[id];

        if(person.id < 6)
        {
            statResults[id] = 1;
        }
        else
        {
            statResults[id] = 0;
        }
    }
    
}

__global__ void analyze_age(Person *people, int *statResults)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {
        Person person = people[id];

        if(person.age != 28)
        {
            statResults[id] = 1;
        }
        else
        {
            statResults[id] = 0;
        }
    }

}

__global__ void analyze_height(Person *people, int *statResults)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {
        Person person = people[id];

        if(person.height != 6)
        {
            statResults[id] = 1;
        }
        else
        {
            statResults[id] = 0;
        }
    }
    
}


int main(void)
{
    int threads = N;
    int blocks = threads/2; // to be more conservative w/blocks: (N + (threads - 1)) / threads
    
    if(blocks > MAX_BLOCKS)
    {
        printf("Data set is too large\nSet cannot exceed %d elements\n", MAX_BLOCKS);
        exit(1);
    }

    cuda_setup(2);

    // CPU variables
    Person *people;
    int idStats[N];
    int ageStats[N];
    int heightStats[N];

    // GPU variables
    Person *dev_people;
    int *dev_idStats;
    int *dev_ageStats;
    int *dev_heightStats;

    // events to track performance time
    float elapsedTime;
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // creates zero-copy memory for buffer (both CPU and GPU point to same memory). A pointer will be given to the GPU later...
    HANDLE_ERROR( hipHostAlloc( (void **) &people, N * sizeof(Person), hipHostMallocWriteCombined | hipHostMallocMapped));

    // allocating GPU memory (GPU only memory)
    HANDLE_ERROR(hipMalloc( (void **) &dev_idStats, N * sizeof(int) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_ageStats, N * sizeof(int) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_heightStats, N * sizeof(int) ));

    /* FILL BUFFER WITH DATA */

    // fill people with some...
    Person person1  = {1, 25, 6};
    Person person2  = {2, 25, 5};
    Person person3  = {3, 25, 6};
    Person person4  = {4, 26, 5};
    Person person5  = {5, 26, 6};
    Person person6  = {6, 26, 5};
    Person person7  = {7, 27, 6};
    Person person8  = {8, 27, 5};
    Person person9  = {9, 27, 6};
    Person person10 = {10, 28, 5};

    Person person11  = {11, 25, 6};
    Person person12  = {12, 25, 5};
    Person person13  = {13, 25, 6};
    Person person14  = {14, 26, 5};
    Person person15  = {15, 26, 6};
    Person person16  = {16, 26, 5};
    Person person17  = {17, 27, 6};
    Person person18  = {18, 27, 5};
    Person person19  = {19, 27, 6};
    Person person20  = {20, 28, 5};

    Person person21  = {21, 25, 6};
    Person person22  = {22, 25, 5};
    Person person23  = {23, 25, 6};
    Person person24  = {24, 26, 5};
    Person person25  = {25, 26, 6};
    Person person26  = {26, 26, 5};
    Person person27  = {27, 27, 6};
    Person person28  = {28, 27, 5};
    Person person29  = {29, 27, 6};
    Person person30  = {30, 28, 5};

    people[0] = person1;
    people[1] = person2;
    people[2] = person3;
    people[3] = person4;
    people[4] = person5;
    people[5] = person6;
    people[6] = person7;
    people[7] = person8;
    people[8] = person9;
    people[9] = person10;

    people[10] = person11;
    people[11] = person12;
    people[12] = person13;
    people[13] = person14;
    people[14] = person15;
    people[15] = person16;
    people[16] = person17;
    people[17] = person18;
    people[18] = person19;
    people[19] = person20;

    people[20] = person21;
    people[21] = person22;
    people[22] = person23;
    people[23] = person24;
    people[24] = person25;
    people[25] = person26;
    people[26] = person27;
    people[27] = person28;
    people[28] = person29;
    people[29] = person30;

    // gives a pointer to the GPU to reference the zero-copy memory
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_people, people, 0));

    // calls to Cuda kernels
    analyze_id<<<blocks, threads>>>(dev_people, dev_idStats);
    analyze_age<<<blocks, threads>>>(dev_people, dev_ageStats);
    analyze_height<<<blocks, threads>>>(dev_people, dev_heightStats);

    // Get the results from the GPU
    HANDLE_ERROR(hipMemcpy(idStats, dev_idStats, N * sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(ageStats, dev_ageStats, N * sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(heightStats, dev_heightStats, N * sizeof(int), hipMemcpyDeviceToHost));

    // make sure everyone is done (only effective with multiple streams...)
    HANDLE_ERROR(hipDeviceSynchronize());

    //stop timing events
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    // cuda cleanup
    hipFree(dev_people);
    hipFree(dev_idStats);
    hipFree(dev_ageStats);
    hipFree(dev_heightStats);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // total results
    int idTotal = 0;
    int ageTotal = 0;
    int heightTotal = 0;

    int x = 0; // loop iterator

    // id totals
    for(x = 0; x < N; x++)
    {
        idTotal += idStats[x];
    }

    // age totals
    for(x = 0; x < N; x++)
    {
        ageTotal += ageStats[x];
    }

    // height totals
    for(x = 0; x < N; x++)
    {
        heightTotal += heightStats[x];
    }

    // print final totals
    printf("\n");
    printf("ID total: %d\n", idTotal);
    printf("Age total: %d\n", ageTotal);
    printf("Height total: %d\n", heightTotal);
    printf("\n");
    printf("blocks used: %d\n", blocks);
    printf("threads used: %d\n", threads);
    printf("threads per block: %d\n", (threads/blocks));
    printf("\n");
    printf("Elapsed time: %3.1f ms\n", elapsedTime);
    printf("\n");
    printf("End of cuda struct test\n");


    return 0;


}

// checks system for cuda compatible devices;
// makes sure the cuda devices have the necessary compute capabilities;
// and picks the most suited device to compute on
// returns an int for status
int cuda_setup(int computeCapability)
{
    int dev;
    //int numOfDevs;

    hipDeviceProp_t prop;

    // assign number of cuda devices to 3rd element in specs array
    //HANDLE_ERROR(hipGetDeviceCount(&numOfDevs));

    // gets current device
    HANDLE_ERROR(hipGetDevice(&dev));

    // create a "pseudo" device w/ desired values and let cuda api pick device that matches
    // memset is just zero-ing out the specified memory
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = computeCapability;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));

    // sets the device w/ the desired paramaters as the device to use
    HANDLE_ERROR(hipSetDevice(dev));

    // allow GPU to zero-copy memory (map)
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

    // get more info about the device
    //HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));

    return 0;
}

