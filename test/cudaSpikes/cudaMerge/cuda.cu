#include "hip/hip_runtime.h"

#include "cuda.cuh"
#include "book.h"


// Declare the Cuda kernels and any Cuda functions

/*
    Searches the Buffer for requests that resulted in a 404.
    Upon finding a 404, a stucture is made out of the request
    containing: who sent the request, when was the request made, what was the request, and if the request was a phpmyadmin injection 
*/
__global__ void analyze_200(char **buffer, Struct200 *results, int *stats)
{ // buffer->requests[id].thing
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < 5)
    {
        printf("GPU: %s\n", buffer[id]);
        stats[id] = id;
    }
}
 

/*
    Takes a Buffer and uses multiple Cuda kernels,
    launched asynchronously, to perform multiple 
    kinds of analysis and statistics on the Buffer's
    contents. Most test will focus on finding potential 
    security intrusions.

    INPUT: Buffer*
    OUTPUT: int, true or false, to determine if analysis was successful
*/
int analyze_data(Buffer *input_buffer)
{
    // localizing buffer values since they will no longer be reliable 
    // after input_buffer is available
    int N = input_buffer->currentSize;
    unsigned int buffer_size = deep_sizeof(input_buffer);//sizeof(input_buffer);//1264696+8;//sizeof(input_buffer);//sizeof(Buffer) * N;//(72+4360)*N;//sizeof(input_buffer->requests[0]);

//############# CPU TESTS ##################################
    int cpuTotal200 = 0;
    int c;
    for(c = 0; c < N; c++)
    {
        if(input_buffer->requests[c].retCode == 200)
            cpuTotal200++;
    }
//##########################################################

    printf("N (input_buffer->currentSize) = %d\n", N);
    printf("buffer_size = %d\n", buffer_size);

    int threads = N;
    int blocks = threads/2; // to be more conservative w/blocks: (N + (threads - 1)) / threads
    fprintf(stderr, "Num threads: %d\n", threads);
    fprintf(stderr, "Num blocks: %d\n", blocks);

    if(blocks > MAX_BLOCKS)
    {
        printf("Data set is too large\nSet cannot exceed %d elements\n", MAX_BLOCKS);
        return FALSE;
    }

    

    // CPU variables
    //Request *cudaBuffer;// = (Buffer *)malloc(sizeof(Buffer));
    char *cudaBuffer[] = {"one", "two", "three", "four", "five"};
    Struct200 results200[N];
    int stats200[N];

    // GPU variables
    //Request *dev_buffer;
    char **dev_buffer;
    Struct200 *dev_results200;
    int *dev_stats200;

    // creates zero-copy memory for buffer (both CPU and GPU point to same memory). A pointer will be given to the GPU later...
    HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer, sizeof(cudaBuffer), hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer[0], 4, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer[1], 4, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer[2], 5, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer[3], 5, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer[4], 5, hipHostMallocWriteCombined | hipHostMallocMapped));
    


    /*int i;
    for(i = 0; i < N; i++)
    {
        cuda_req_init(&cudaBuffer[i]);
    }*/

    /*memcpy(cudaBuffer, input_buffer->requests, sizeof(Request) * N);
    int j;
    for(j = 0; j < N; j++)
    {
        printf("CPU-Host: %s\tCPU-Return Code: %d\n", cudaBuffer[j].host, cudaBuffer[j].retCode);
    }*/

    // allocating GPU memory (GPU only memory)
    HANDLE_ERROR(hipMalloc( (void **) &dev_results200, N * sizeof(Struct200) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_stats200, N * sizeof(int) ));

    // page-locking output buffers (pin host memory for streams)
    /*HANDLE_ERROR(hipHostAlloc( (void **) &results200, N * sizeof(Struct200), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc( (void **) &stats200, N * sizeof(int), hipHostMallocDefault));*/

    /* FILL BUFFER WITH DATA */

    // gives a pointer to the GPU to reference the zero-copy memory
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_buffer, *cudaBuffer, 0));

    // calls to Cuda kernels, note streams have been added
    analyze_200<<< blocks, threads >>>(dev_buffer, dev_results200, dev_stats200);

    // Get the results from the GPU
    HANDLE_ERROR(hipMemcpy(results200, dev_results200, N * sizeof(Struct200), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(stats200, dev_stats200, N * sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipDeviceSynchronize());

    /* Print output for testing */
    int total200 = 0;

    int x;
    for(x = 0; x < N; x++)
    {
        total200 += stats200[x];

        fprintf(stderr, "200[%d] = %d\n", x, stats200[x]);
    }

    printf("CPU total 200: %d\n", cpuTotal200);
    printf("Total 200s: %d\n", total200);

    // cuda cleanup
    hipFree(dev_buffer);
    hipFree(dev_results200);
    hipFree(dev_stats200);

    hipFree(cudaBuffer); // zero-copy

    /* Send results to files?? */

    return TRUE;
}

int cuda_setup(int computeCapability)
{
    int dev;
    //int numOfDevs;

    hipDeviceProp_t prop;

    // assign number of cuda devices to 3rd element in specs array
    //HANDLE_ERROR(hipGetDeviceCount(&numOfDevs));

    // gets current device
    HANDLE_ERROR(hipGetDevice(&dev));

    // create a "pseudo" device w/ desired values and let cuda api pick device that matches
    // memset is just zero-ing out the specified memory
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = computeCapability;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));

    // sets the device w/ the desired paramaters as the device to use
    HANDLE_ERROR(hipSetDevice(dev));

    // allow GPU to zero-copy memory (map)
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

    // get more info about the device
    //HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));

    return 0;
}

// Copies strings in cuda code
// Returns: pointer to dest
char * cuda_strcpy(char *dest, const char *source)
{
    int length = strlen(source);
    dest = (char *) malloc(length + 1);

    int x = 0;
    for(x = 0; x < length; x++)
    {
        dest[x] = source[x];
    }
    dest[length + 1] = '\0';

    return dest;

    
}

// Compares strings in cuda code
// Returns: 0 if equal, -1 if not
int cuda_strcmp(char *str1, char *str2)
{
    unsigned int str1Len = sizeof(str1);
    unsigned int str2Len = sizeof(str2);

    if(str1Len != str2Len)
    {
        return -1;
    }
    else
    {
        int x;
        for(x = 0; x < str1Len; x++)
        {
            if(str1[x] != str2[x])
            {
                return -1;
            }
        }
    }

    return 0;

}

unsigned int deep_sizeof(Buffer *buffer)
{
    unsigned int size = 0;
    size += sizeof(Buffer);
    size += sizeof(Request) * buffer->currentSize;

    int x;
    for(x = 0; x < buffer->currentSize; x++)
    {
        size += sizeof(buffer->requests[x]);
    }

    return size;
}

// Performs Zero-Copy
int cuda_buffer_init(Buffer * buffer){
    int i;
    //buffer->requests = (Request *)malloc((BUFFER_SIZE) * sizeof(Request));
    HANDLE_ERROR( hipHostAlloc( (void **) &buffer, sizeof(Buffer), hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &buffer->requests, sizeof(Request) * BUFFER_SIZE, hipHostMallocWriteCombined | hipHostMallocMapped));
    for (i = 0; i < BUFFER_SIZE; i++)
    {
        if (cuda_req_init(&buffer->requests[i]))
        {
            //debug_write("request struct allocation error!\n");
            return 1;
        }
    }
    return 0;
}

int
cuda_req_init(Request *request)
{

    HANDLE_ERROR( hipHostAlloc( (void **) &request->host, sizeof(char) * 1000, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &request->clientId, sizeof(char) * 50, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &request->userId, sizeof(char) * 150, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &request->time, sizeof(char) * 100, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &request->req, sizeof(char) * 2000, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &request->referer, sizeof(char) * 30, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &request->userAgent, sizeof(char) * 30, hipHostMallocWriteCombined | hipHostMallocMapped));
    //HANDLE_ERROR( hipHostAlloc( (void **) &request->retCode, sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped));
    //HANDLE_ERROR( hipHostAlloc( (void **) &request->dataSize, sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped));

    return 0;
}