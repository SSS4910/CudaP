#include "hip/hip_runtime.h"

#include "cuda.cuh"
#include "book.h"


// Declare the Cuda kernels and any Cuda functions
__global__ void analyze_404(Buffer *buffer, Struct404 *results, int *stats)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < buffer->currentSize)
    {
        if( buffer->requests[id].retCode == 404)
        {
           /* cuda_strcpy(results[id].host, buffer->requests[id]->host);
            cuda_strcpy(results[id].req, buffer->requests[id]->req);
            cuda_strcpy(results[id].time, buffer->requests[id]->time);

            if(cuda_strcmp(buffer->requests[id]->req, "/PHPMYADMIN/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/PMA/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/PMA2005/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/SSLMySQLAdmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/SQL/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/admin/phpmyadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/admin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/bbs/data/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpadmindb/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/admin/pma/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpanelmysql/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpanelphpmyadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpanelsql/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpphpmyadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/db/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/dbadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/myadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysql-admin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysql/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysqladmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysqladminconfig/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysqlmanager/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else
            {
                results[id].is_injection = FALSE;
            }*/

            //if(results[id].is_injection)
           // {
                stats[id] = 1;
            //}

        }
        else
        {
            stats[id] = 0;
        }
    }
    
}

__global__ void analyze_200(Buffer *buffer, Struct200 *results, int *stats)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < buffer->currentSize)
    {
        if(buffer->requests[id].retCode == 200)
        {
            stats[id] = 1;
        }
        else
        {
            stats[id] = 0;
        }
    }

}

/*__global__ void analyze_height(Buffer *buffer, int *statResults)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {

    }
    
}*/

int analyze_data(Buffer *input_buffer)
{
    // localizing buffer values since they will no longer be reliable 
    // after input_buffer is available
    int N = input_buffer->currentSize;
    unsigned int buffer_size = sizeof(input_buffer);

    int threads = N;
    int blocks = threads/2; // to be more conservative w/blocks: (N + (threads - 1)) / threads
    
    if(blocks > MAX_BLOCKS)
    {
        printf("Data set is too large\nSet cannot exceed %d elements\n", MAX_BLOCKS);
        exit(1);
    }

    cuda_setup(COMPUTE_ABILITY);

    // CPU variables
    Buffer *cudaBuffer;
    Struct404 results404[N];
    Struct200 results200[N];
    int stats404[N];
    int stats200[N];

    // GPU variables
    Buffer *dev_buffer;
    Struct404 *dev_results404;
    Struct200 *dev_results200;
    int *dev_stats404;
    int *dev_stats200;

    // events to track performance time
    float elapsedTime;
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // creates zero-copy memory for buffer (both CPU and GPU point to same memory). A pointer will be given to the GPU later...
    HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer, buffer_size, hipHostMallocWriteCombined | hipHostMallocMapped));

    // fill the zero-copy memory with data from input_buffer
    cudaBuffer = (Buffer *)memcpy(&cudaBuffer, &input_buffer, buffer_size);

    // lets main know the buffer is free
    input_buffer->available = TRUE;

    //Declare streams
    hipStream_t stream0;
    hipStream_t stream1;
    //hipStream_t stream2;

    //create steams
    HANDLE_ERROR(hipStreamCreate(&stream0));
    HANDLE_ERROR(hipStreamCreate(&stream1));
    //HANDLE_ERROR(hipStreamCreate(&stream2));

    // allocating GPU memory (GPU only memory)
    HANDLE_ERROR(hipMalloc( (void **) &dev_results404, N * sizeof(Struct404) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_stats404, N * sizeof(int) ));

    HANDLE_ERROR(hipMalloc( (void **) &dev_results200, N * sizeof(Struct200) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_stats200, N * sizeof(int) ));
    
    //HANDLE_ERROR(hipMalloc( (void **) &dev_heightStats, N * sizeof(int) ));

    // page-locking output buffers (pin host memory for streams)
    HANDLE_ERROR(hipHostAlloc( (void **) &results404, N * sizeof(Struct404), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc( (void **) &stats404, N * sizeof(int), hipHostMallocDefault));

    HANDLE_ERROR(hipHostAlloc( (void **) &results200, N * sizeof(Struct200), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc( (void **) &stats200, N * sizeof(int), hipHostMallocDefault));
    
    //HANDLE_ERROR(hipHostAlloc( (void **) &heightStats, N * sizeof(int), hipHostMallocDefault));

    /* FILL BUFFER WITH DATA */

    // gives a pointer to the GPU to reference the zero-copy memory
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_buffer, cudaBuffer, 0));

    // calls to Cuda kernels, note streams have been added
    analyze_404<<< blocks, threads, 0, stream0 >>>(dev_buffer, dev_results404, dev_stats404);
    analyze_200<<< blocks, threads, 0, stream1 >>>(dev_buffer, dev_results200, dev_stats200);
    //analyze_height<<< blocks, threads, 0, stream2 >>>(dev_buffer, dev_heightStats);

    // Get the results from the GPU
    HANDLE_ERROR(hipMemcpyAsync(results404, dev_results404, N * sizeof(Struct404), hipMemcpyDeviceToHost, stream0));
    HANDLE_ERROR(hipMemcpyAsync(stats404, dev_stats404, N * sizeof(int), hipMemcpyDeviceToHost, stream0));

    HANDLE_ERROR(hipMemcpyAsync(results200, dev_results200, N * sizeof(Struct200), hipMemcpyDeviceToHost, stream1));
    HANDLE_ERROR(hipMemcpyAsync(stats200, dev_stats200, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
    //HANDLE_ERROR(hipMemcpyAsync(heightStats, dev_heightStats, N * sizeof(int), hipMemcpyDeviceToHost, stream2));

    // make sure everyone is done
    HANDLE_ERROR(hipStreamSynchronize(stream0));
    HANDLE_ERROR(hipStreamSynchronize(stream1));
    //HANDLE_ERROR(hipStreamSynchronize(stream2));

    HANDLE_ERROR(hipDeviceSynchronize());

    //stop timing events
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    /* Print output for testing */
    int total404 = 0;
    int total200 = 0;

    int x;
    for(x = 0; x < N; x++)
    {
        total404 += stats404[x];
        total200 += stats200[x];

        /*if(results404[x] != NULL)
            printf("%s : %s\n", results404[x]->host, results404[x]->req);*/
    }

    printf("Total 404s: %d\n", total404);
    printf("Total 200s: %d\n", total404);

    // cuda cleanup
    hipFree(dev_buffer);
    hipFree(dev_results404);
    hipFree(dev_stats404);
    hipFree(dev_results200);
    hipFree(dev_stats200);
    hipFree(results404);
    hipFree(stats404);
    hipFree(results200);
    hipFree(stats200);
    //hipFree(dev_heightStats);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    //hipStreamDestroy(stream2);

    /* Send results to files?? */


    return 0;
}

int cuda_setup(int computeCapability)
{
    int dev;
    //int numOfDevs;

    hipDeviceProp_t prop;

    // assign number of cuda devices to 3rd element in specs array
    //HANDLE_ERROR(hipGetDeviceCount(&numOfDevs));

    // gets current device
    HANDLE_ERROR(hipGetDevice(&dev));

    // create a "pseudo" device w/ desired values and let cuda api pick device that matches
    // memset is just zero-ing out the specified memory
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = computeCapability;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));

    // sets the device w/ the desired paramaters as the device to use
    HANDLE_ERROR(hipSetDevice(dev));

    // allow GPU to zero-copy memory (map)
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

    // get more info about the device
    //HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));

    return 0;
}

// Copies strings in cuda code
// Returns: pointer to dest
char * cuda_strcpy(char *dest, const char *source)
{
    int length = strlen(source);
    dest = (char *) malloc(length + 1);

    int x = 0;
    for(x = 0; x < length; x++)
    {
        dest[x] = source[x];
    }
    dest[length + 1] = '\0';

    return dest;

    
}

// Compares strings in cuda code
// Returns: 0 if equal, -1 if not
int cuda_strcmp(char *str1, char *str2)
{
    unsigned int str1Len = sizeof(str1);
    unsigned int str2Len = sizeof(str2);

    if(str1Len != str2Len)
    {
        return -1;
    }
    else
    {
        int x;
        for(x = 0; x < str1Len; x++)
        {
            if(str1[x] != str2[x])
            {
                return -1;
            }
        }
    }

    return 0;

}