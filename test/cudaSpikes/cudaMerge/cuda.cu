#include "hip/hip_runtime.h"

#include "cuda.cuh"
#include "book.h"


// Declare the Cuda kernels and any Cuda functions

/*
    Searches the Buffer for requests that resulted in a 404.
    Upon finding a 404, a stucture is made out of the request
    containing: who sent the request, when was the request made, what was the request, and if the request was a phpmyadmin injection 
*/
__global__ void analyze_404(Buffer *buffer, Struct404 *results, int *stats)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < 15)//buffer->currentSize)
    {
        if( buffer->requests[id].retCode == 404)
        {
            stats[id] = 1;
           /* cuda_strcpy(results[id].host, buffer->requests[id]->host);
            cuda_strcpy(results[id].req, buffer->requests[id]->req);
            cuda_strcpy(results[id].time, buffer->requests[id]->time);

            if(cuda_strcmp(buffer->requests[id]->req, "/PHPMYADMIN/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/PMA/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/PMA2005/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/SSLMySQLAdmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/SQL/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/admin/phpmyadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/admin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/bbs/data/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpadmindb/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/admin/pma/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpanelmysql/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpanelphpmyadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpanelsql/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/cpphpmyadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/db/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/dbadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/myadmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysql-admin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysql/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysqladmin/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysqladminconfig/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else if(cuda_strcmp(buffer->requests[id]->req, "/mysqlmanager/scripts/setup.php") == 0)
            {
                results[id].is_injection = TRUE;
            }
            else
            {
                results[id].is_injection = FALSE;
            }

            //if(results[id].is_injection)
           // {
                stats[id] = 1;
            //}*/

        }
        else
        {
            stats[id] = 0;
        }
    }
    else
    {
        stats[id] = buffer->requests[id].retCode;
    }
    
}

__global__ void analyze_200(Buffer *buffer, Struct200 *results, int *stats)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < 15)//buffer->currentSize)
    {
        if(buffer->requests[id].retCode == 200)
        {
            stats[id] = 1;
        }
        else
        {
            stats[id] = 0;
        }
    }
    else
    {
        stats[id] = buffer->requests[id].retCode;
    }
}

/*__global__ void analyze_height(Buffer *buffer, int *statResults)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {

    }
    
}*/

/*
    Takes a Buffer and uses multiple Cuda kernels,
    launched asynchronously, to perform multiple 
    kinds of analysis and statistics on the Buffer's
    contents. Most test will focus on finding potential 
    security intrusions.

    INPUT: Buffer*
    OUTPUT: int, true or false, to determine if analysis was successful
*/
int analyze_data(Buffer *input_buffer)
{
    // localizing buffer values since they will no longer be reliable 
    // after input_buffer is available
    int N = input_buffer->currentSize;
    unsigned int buffer_size = 1264696+8;//sizeof(input_buffer);//sizeof(Buffer) * N;//(72+4360)*N;//sizeof(input_buffer->requests[0]);

    int cpuTotal404 = 0;
    int cpuTotal200 = 0;
    fprintf(stderr, "Calculating CPU totals...\n");
    int c;
    for(c = 0; c < N; c++)
    {
        if(input_buffer->requests[c].retCode == 404)
            cpuTotal404++;

        if(input_buffer->requests[c].retCode == 200)
            cpuTotal200++;
    }

    printf("N (input_buffer->currentSize) = %d\n", N);
    printf("buffer_size = %d\n", buffer_size);

    int threads = N;
    int blocks = threads/2; // to be more conservative w/blocks: (N + (threads - 1)) / threads
    fprintf(stderr, "Num threads: %d\n", threads);
    fprintf(stderr, "Num blocks: %d\n", blocks);

    if(blocks > MAX_BLOCKS)
    {
        printf("Data set is too large\nSet cannot exceed %d elements\n", MAX_BLOCKS);
        return FALSE;
    }

    //cuda_setup(COMPUTE_ABILITY);

    // CPU variables
    //Buffer *cudaBuffer;
    Struct404 results404[N];
    Struct200 results200[N];
    int stats404[N];
    int stats200[N];

    // GPU variables
    Buffer *dev_buffer;
    Struct404 *dev_results404;
    Struct200 *dev_results200;
    int *dev_stats404;
    int *dev_stats200;

    // events to track performance time
    float elapsedTime;
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    fprintf(stderr, "Before zero-copy\n" );

    // creates zero-copy memory for buffer (both CPU and GPU point to same memory). A pointer will be given to the GPU later...
    //HANDLE_ERROR( hipHostAlloc( (void **) &cudaBuffer, buffer_size, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR( hipHostAlloc( (void **) &input_buffer, buffer_size, hipHostMallocWriteCombined | hipHostMallocMapped));
    fprintf(stderr, "After zero-copy\n" );

    // fill the zero-copy memory with data from input_buffer
    //memcpy(&cudaBuffer, &input_buffer, buffer_size);
    //fprintf(stderr, "After buffer memcpy\n" );
    // lets main know the buffer is free
    //input_buffer->available = TRUE;

    //Declare streams
    hipStream_t stream0;
    hipStream_t stream1;
    //hipStream_t stream2;

    //create steams
    HANDLE_ERROR(hipStreamCreate(&stream0));
    HANDLE_ERROR(hipStreamCreate(&stream1));
    //HANDLE_ERROR(hipStreamCreate(&stream2));

    // allocating GPU memory (GPU only memory)
    HANDLE_ERROR(hipMalloc( (void **) &dev_results404, N * sizeof(Struct404) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_stats404, N * sizeof(int) ));

    HANDLE_ERROR(hipMalloc( (void **) &dev_results200, N * sizeof(Struct200) ));
    HANDLE_ERROR(hipMalloc( (void **) &dev_stats200, N * sizeof(int) ));

    // page-locking output buffers (pin host memory for streams)
    HANDLE_ERROR(hipHostAlloc( (void **) &results404, N * sizeof(Struct404), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc( (void **) &stats404, N * sizeof(int), hipHostMallocDefault));

    HANDLE_ERROR(hipHostAlloc( (void **) &results200, N * sizeof(Struct200), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc( (void **) &stats200, N * sizeof(int), hipHostMallocDefault));

    /* FILL BUFFER WITH DATA */

    // gives a pointer to the GPU to reference the zero-copy memory
    //HANDLE_ERROR(hipHostGetDevicePointer(&dev_buffer, cudaBuffer, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_buffer, input_buffer, 0));

    // calls to Cuda kernels, note streams have been added
    analyze_404<<< blocks, threads, 0, stream0 >>>(dev_buffer, dev_results404, dev_stats404);
    analyze_200<<< blocks, threads, 0, stream1 >>>(dev_buffer, dev_results200, dev_stats200);
    //analyze_height<<< blocks, threads, 0, stream2 >>>(dev_buffer, dev_heightStats);

    // Get the results from the GPU
    HANDLE_ERROR(hipMemcpyAsync(results404, dev_results404, N * sizeof(Struct404), hipMemcpyDeviceToHost, stream0));
    HANDLE_ERROR(hipMemcpyAsync(stats404, dev_stats404, N * sizeof(int), hipMemcpyDeviceToHost, stream0));

    HANDLE_ERROR(hipMemcpyAsync(results200, dev_results200, N * sizeof(Struct200), hipMemcpyDeviceToHost, stream1));
    HANDLE_ERROR(hipMemcpyAsync(stats200, dev_stats200, N * sizeof(int), hipMemcpyDeviceToHost, stream1));

    // make sure everyone is done
    HANDLE_ERROR(hipStreamSynchronize(stream0));
    HANDLE_ERROR(hipStreamSynchronize(stream1));
    //HANDLE_ERROR(hipStreamSynchronize(stream2));

    HANDLE_ERROR(hipDeviceSynchronize());

    //stop timing events
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    /* Print output for testing */
    int total404 = 0;
    int total200 = 0;
    
    /*int x = 0;
    for(x = 0; x < N; x++)
    {
        if(input_buffer->requests[x].retCode == 404)
        {
            total404++;
            fprintf(stderr, "    %s %d\n", input_buffer->requests[x].host, input_buffer->requests[x].retCode);
        }
        else if(input_buffer->requests[x].retCode == 200)
        {
            total200++;
            fprintf(stderr, "%s %d\n", input_buffer->requests[x].host, input_buffer->requests[x].retCode);
        }
    }*/



    int x;
    for(x = 0; x < N; x++)
    {
        total404 += stats404[x];
        total200 += stats200[x];

        fprintf(stderr, "404[%d] = %d\n", x, stats404[x]);
        fprintf(stderr, "\t\t\t200[%d] = %d\n", x, stats200[x]);
    }

    printf("CPU total 404: %d\n", cpuTotal404);
    printf("CPU total 200: %d\n", cpuTotal200);
    printf("Total 404s: %d\n", total404);
    printf("Total 200s: %d\n", total200);

    // cuda cleanup
    hipFree(dev_buffer);
    hipFree(dev_results404);
    hipFree(dev_stats404);
    hipFree(dev_results200);
    hipFree(dev_stats200);
    hipFree(results404);
    hipFree(stats404);
    hipFree(results200);
    hipFree(stats200);

    hipFree(input_buffer);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    //hipStreamDestroy(stream2);

    /* Send results to files?? */


    return TRUE;
}

int cuda_setup(int computeCapability)
{
    int dev;
    //int numOfDevs;

    hipDeviceProp_t prop;

    // assign number of cuda devices to 3rd element in specs array
    //HANDLE_ERROR(hipGetDeviceCount(&numOfDevs));

    // gets current device
    HANDLE_ERROR(hipGetDevice(&dev));

    // create a "pseudo" device w/ desired values and let cuda api pick device that matches
    // memset is just zero-ing out the specified memory
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = computeCapability;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));

    // sets the device w/ the desired paramaters as the device to use
    HANDLE_ERROR(hipSetDevice(dev));

    // allow GPU to zero-copy memory (map)
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

    // get more info about the device
    //HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));

    return 0;
}

// Copies strings in cuda code
// Returns: pointer to dest
char * cuda_strcpy(char *dest, const char *source)
{
    int length = strlen(source);
    dest = (char *) malloc(length + 1);

    int x = 0;
    for(x = 0; x < length; x++)
    {
        dest[x] = source[x];
    }
    dest[length + 1] = '\0';

    return dest;

    
}

// Compares strings in cuda code
// Returns: 0 if equal, -1 if not
int cuda_strcmp(char *str1, char *str2)
{
    unsigned int str1Len = sizeof(str1);
    unsigned int str2Len = sizeof(str2);

    if(str1Len != str2Len)
    {
        return -1;
    }
    else
    {
        int x;
        for(x = 0; x < str1Len; x++)
        {
            if(str1[x] != str2[x])
            {
                return -1;
            }
        }
    }

    return 0;

}