#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <string.h>

#include "core.h"
#include "parser.h"
#include "debug.h"
#include "cuda.cuh"

//this shouldn't be here, but...
int
parse_line(char *, Request *);



/*
 * FUNCTION: main
 * --------------
 * the entry point of the application
 *
 * argc: the number of command line arguments
 * argv: the command line arguments, kept as a pointer to a string
 *
 * returns: 0 on successful completion, -1 on failure
 */
int
main(int argc, char** argv){
    int i;
    int err;
    int lineNum = 0;
    char * logline;
    FILE * logfile;
    Buffer buffer;

    //getopt()
    //open_debug_file();

    fprintf(stdout, "Nothing has happened yet\n");

    // choose cuda device
    cuda_setup(COMPUTE_ABILITY);

    //debug_write("Opening access.log file\n");
    if ((logfile = fopen("../../../../access.log", "r")) == NULL)
    {
        //debug_write("access.log not found, aborting!\n");
        printf("access.log not found, aborting!\n");
        return -1;
    }

    fprintf(stdout, "Server log was opened\n");

    //debug_write("Allocating memory for buffer\n");
    err = buffer_init(&buffer);
    if (err)
    {
        //debug_write("buffer allocation failure, aborting!\n");
        return -1;
    }
    buffer.available = TRUE;
    fprintf(stdout, "Buffer created\n");
    /*

    strcpy(buffer.requests[0].host, "127.0.0.1");

    printf("%s\n", buffer.requests[0].host);

    req_null(&buffer.requests[0]);

    printf("%s\n", buffer.requests[0].host);

    */
    fprintf(stdout, "Doing the loop\n");
    while (!feof(logfile))
    {
        buffer.currentSize = 0;
        for (i = 0; i < BUFFER_SIZE; i++)
        {
            //insure that the structure is empty
            req_null(&buffer.requests[i]);
            //read a line
            logline = log_readline(logfile);
            if ((logline == (char *) NULL) && feof(logfile))
            {
                //printf("hit eof\n");
                break;
            }
            lineNum++;
            //printf("%s\n", logline);
            //parse line and add it to the buffer
            err = parse_line(logline, &buffer.requests[i]);
            if (err)
            {
                //printf("parse error on line %d\n", lineNum);
            }
            buffer.currentSize++;
            free(logline);
            /*printf("%s\n", buffer.requests[i].host);
            printf("%s\n", buffer.requests[i].clientId);
            printf("%s\n", buffer.requests[i].userId);
            printf("%s\n", buffer.requests[i].time);
            printf("%s\n", buffer.requests[i].req);
            printf("%d\n", buffer.requests[i].retCode);
            printf("%d\n", buffer.requests[i].dataSize);
            fflush(stdout);*/
            //fprintf(stdout, "Test in main... %d\n", (&buffer.requests[i].host) -1 );

        }

        // call to cuda analysis
        //fprintf(stderr, "Size %d\n", (int)sizeof(buffer));
        int cudaSucces = analyze_data(&buffer);
        if(!cudaSucces)
        {
            printf("Cuda Module Failed to Properly Execute\n");
        }
        fprintf(stdout, "Forced end\n");
        exit(0);
    }

    fprintf(stdout, "Loop ended\n");

    //cleanup
    buffer_free(&buffer);
    //debug_write("Freeing memory for line buffer\n");

    fclose(logfile);
    //debug_write("Closing access.log file...\n");
    //close_debug_file();
    return 0;
}


/*
 * FUNCTION: buffer_init
 * -----------------------
 * initialize buffer to store each line of the log
 *
 * return:
 *  a pointer to a buffer structure on success, NULL on failure
 */
int 
buffer_init(Buffer * buffer){
    int i;
    buffer->requests = (Request *)malloc((BUFFER_SIZE) * sizeof(Request));
    for (i = 0; i < BUFFER_SIZE; i++)
    {
        if (req_init(&buffer->requests[i]))
        {
            //debug_write("request struct allocation error!\n");
            return 1;
        }
    }
    return 0;
}

/*
 * FUNCTION: buffer_free
 * ---------------------
 * frees memory allocated to buffer
 */
void
buffer_free(Buffer * buffer){
    int i = 0;
    for (i = 0; i < BUFFER_SIZE; i++)
    {
        req_free(&buffer->requests[i]);
    }
    //debug_write("free requests pointer\n");
    free(buffer->requests);
}

/*
 * FUNCTION: req_init
 * ------------------
 * allocates memory for a request
 *
 * request: Request structure requiring memory
 *
 * return:
 *  0 on success
 */
int
req_init(Request *request)
{
    //debug_write("allocating request structure field memory!\n");
    request->host = (char *) malloc(1000 * sizeof(char));
    request->clientId = (char *) malloc(50 * sizeof(char));
    request->userId = (char *) malloc(150 * sizeof(char));
    request->time = (char *) malloc(100 * sizeof(char));
    request->req = (char *) malloc(2000 * sizeof(char));
    request->referer = (char *) malloc(30 * sizeof(char));
    request->userAgent = (char *) malloc(30 * sizeof(char));
    return 0;
}

/*
 * FUNCTION: req_null
 * ------------------
 * tombstones a request structure
 *
 * return:
 *  0 on success
 */
int
req_null(Request *request)
{
    strcpy(request->host, "~");
    strcpy(request->clientId, "~");
    strcpy(request->userId, "~");
    strcpy(request->time, "~");
    strcpy(request->req, "~");
    request->retCode = -1;
    request->dataSize = -1;
    strcpy(request->referer, "~");
    strcpy(request->userAgent, "~");
    return 0;
}


/*
 * FUNCTION: req_free
 * ------------------
 * frees individual fields of provided request
 *
 * request: Request structure to free
 */
void
req_free(Request *request)
{
    //debug_write("freeing request structure field memory\n");
    free(request->host);
    free(request->clientId);
    free(request->userId);
    free(request->time);
    free(request->req);
    free(request->referer);
    free(request->userAgent);
}

/*
 * FUNCTION: log_readline
 * ----------------------
 * reads next line from provided log file
 *
 * logfile: the log file pointer
 *
 * return:
 *  a newline terminated string containing the next line from file
 */
char *
log_readline(FILE * logfile){
    char * line = (char *)malloc((MAX_LINE_LENGTH) * sizeof(char));
    if (fgets(line, MAX_LINE_LENGTH, logfile) != NULL)
    {
        return line;
    }
    return (char *)NULL;
}






// should not be here
int
parse_line(char * line, Request *request){
    //printf("%s \n",line);

    /*static int const HOSTSIZE = 1000;
    static int const CLIENTIDSIZE = 50;
    static int const USERIDSIZE = 150;
    static int const TIMESIZE = 100;
    static int const REQSIZE = 2000;
    static int const CODESIZE = 30;
    static int const SIZESIZE = 30;
    static int const REFERERSIZE = 30;
    static int const USERAGENTSIZE = 30;*/
    
        //counters
    int n = 0;
    int j = 0;
    int i = 0;
    int h = 0;
    int k = 0;

    

        // alocate memory and create the Request that will be returned
    
    //Request * outRequest;
    //outRequest = malloc(sizeof(Request*));
    /*outRequest = malloc((sizeof(char)* 
                        (
                        HOSTSIZE+
                        CLIENTIDSIZE+
                        USERIDSIZE+
                        TIMESIZE+
                        REQSIZE+
                        REFERERSIZE+
                        USERAGENTSIZE))
                    + (sizeof(int) * 20));
    
    outRequest->host = (char *)malloc(HOSTSIZE * sizeof(char));    
      
    outRequest->clientId = (char *)malloc(CLIENTIDSIZE * sizeof(char)); 
     
    outRequest->userId = (char *)malloc((USERIDSIZE) * sizeof(char));  
    
    outRequest->time = (char *)malloc(TIMESIZE * sizeof(char));  
    
    outRequest->req = (char *)malloc((REQSIZE) * sizeof(char));  
    */
    char* retCode;
    retCode = (char *)malloc((30) * sizeof(char));  
       
    char* retSize;
    retSize = (char *)malloc((30) * sizeof(char));  
    /*   
    outRequest->referer = (char *)malloc((REFERERSIZE) * sizeof(char));  

    outRequest->userAgent = (char *)malloc(USERAGENTSIZE * sizeof(char));   
    */
    int flag = 0;
    


        // loop for every character in the input line 
    for(i =0; i < strlen(line); i++)
    {

            //fflush(stdout);

            //this parses and breaks down the line into fields, each case is a different 
        switch(n)
        {

                //field one 
                //      (%h) host field
                //          this is either and IP or a website
                //
                //          terminated by ' '
            case 0:
                    //printf(" In field one \n");
                
                if (line[i] == ' ')
                {
                    n = 1;
                    request->host[j] = '\0';
                    j =0;
                }
                else
                {
                    request->host[j] = line[i];
                    j++;
                }
                break;

                //field two 
                //      (%l) RFC
                //          request 1413
                //
                //          terminated by ' '
            case 1:
                    //printf(" In field two \n");

                if(line[i] == ' ')
                {
                    n = 2;
                    request->clientId[j] = '\0';
                    j =0;
                }
                else
                {
                    request->clientId[j] = line[i];
                    j++;
                }
                break;

                //field three
                //      (%u)    user id
                //      
                //      terminated by a ' '
            case 2:
                //printf(" In field three \n");

                if(line[i] == ' ')
                {
                    n = 3;
                    request->userId[j] = '\0';
                    j = 0;
                }
                else
                {
                    request->userId[j] = line[i];
                    j++;
                }
                break;

                //field four 
                //      (%t) time
                //      this is the time field 
                //      
                //      [day/month/year:hour:minute:second zone]
                //      day = 2*digit
                //      month = 3*letter
                //      year = 4*digit
                //      hour = 2*digit
                //      minute = 2*digit
                //      second = 2*digit
                //      zone = (`+' | `-') 4*digit
                //
                //      terminated by a ']'
            case 3:
                    //printf(" In field four \n");
                if(flag == 1)
                {
                    n = 4;
                    flag = 0;
                    request->time[j] = '\0';
                    j = 0;

                }
                else if(line[i] == ']')
                {
                    flag = 1;
                    request->time[j] = line[i];
                    j++;
                }
                else
                {
                    request->time[j] = line[i];
                    j++;
                }
                break;
           
                //field five
                //      (\"%r\") request line
                //
                //      altho the program begins and ends with a '"' you cannot just assume that
                //      you have hit the end of the feild becuase you hit a second '"' this is why
                //      field six has to check and make sure field five(request line) has ended
                //
                //      terminated by 2nd '"' 
                //      
                //      PROBLAMATIC

            case 4:
                    //printf(" In field five \n");

                if(flag == 2)
                {
                    if (line[i] != ' ')
                    {
                        flag = 1;
                        n = 4;
                        request->req[j] = line[i];
                        j++;
                    }
                    else
                    {
                        flag = 0;
                        n = 5;
                        h = 0;
                        request->req[j] = '\0';
                    }
                }
                else if( line[i] == '"')
                {
                    request->req[j] = line[i];
                    j++;
                    flag++;
                }
                else
                {
                    request->req[j] = line[i];
                    j++;
                }
                break;

                //field six 
                //      (%>s) status code
                //
                //      this is a relatively simple field that usually returns 200 or 404
                //          this also checks to make sure the arnt still looking at the request 
                //          line because of a false positve
                //
                //          terminated by a ' '
                //
            case 5:
                    //printf(" In field six \n");

                if(line[i] == ' ')
                {
                    retCode[h] = '\0';
                    request->retCode = atoi(retCode);
                    if(request->retCode == 0)
                    {
                        //n = 5;
                        for(k = 0; k < h; k++)
                        {
                            request->req[j] = retCode[k];
                            j++;
                        }
                        h = 0;
                    }
                    else
                    {
                        n = 6;
                        h = 0;
                        j = 0;
                    }
                }
                else
                {
                    
                    retCode[h] = line[i];
                    h++;
                }
                break;

                //  feild seven
                //      (%b) size of data
                //
                //      this is just how much data was returned 
                //
                //      terminated by a ' ' or a '\n' for "short" logs
                //
            case 6:
                    //printf(" In field seven \n");

                if(line[i] == ' ' || line[i] == '\n')
                {
                    n = 7;
                    retSize[h] = '\0';
                    h = 0;
                    request->dataSize = atoi(retSize);
                }
                else
                {
                    retSize[h] = line[i];
                    h++;
                }
                break;

                //COMBINED FORMAT LOG DATA STILL NEEDED

        }   


    }

    //printf("   Inside: H:%s R:%s A:%s T:%s C:%s c:%d S:%d \n \n",outRequest->host, outRequest->clientId, outRequest->userId, outRequest->time, outRequest->req, outRequest->httpReturnCode, outRequest->dataSize );
    
    
    /*free(outRequest.host);
    free(outRequest.rfc);
    free(outRequest.auth);
    free(outRequest.time);
    free(outRequest.req);*/
    free(retCode);
    free(retSize);
    /*free(outRequest.referer);
    free(outRequest.userAgent);*/

    return 0;
}
